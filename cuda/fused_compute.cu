
#include <hip/hip_runtime.h>
#ifdef FMOE_USE_NCCL

#include <vector>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include "fused_compute.cuh"

std::vector<torch::Tensor> _fused_forward(
        torch::Tensor input_buf,
        torch::Tensor weight1,
        torch::Tensor weight2,
        torch::Tensor local_expert_count,
        torch::Tensor global_expert_count,
        long global_batch_size,
        long n_workers, bool has_bias) {
    const auto num_expert = local_expert_count.size(0) / n_workers;
    const auto d_hidden = weight1.size(1);
    const auto d_model = weight1.size(2);

    auto smgr = getCudaStreamManager(input_buf.device().index());

    auto global_input_buf = input_buf.new_empty({global_batch_size, d_model});
    auto global_middle_buf = input_buf.new_empty({global_batch_size, d_hidden});
    auto global_output_buf = input_buf.new_empty({global_batch_size, d_model});
    auto output_buf = input_buf.new_empty({input_buf.size(0), d_model});

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), 
            "fmoe_cuda_fused_forward", ([&] {
        fmoe_cuda_fused_forward_impl(
            input_buf.data_ptr<scalar_t>(),
            weight1.data_ptr<scalar_t>(),
            weight2.data_ptr<scalar_t>(),

            global_input_buf.data_ptr<scalar_t>(),
            global_middle_buf.data_ptr<scalar_t>(),
            global_output_buf.data_ptr<scalar_t>(),
            output_buf.data_ptr<scalar_t>(),

            local_expert_count.data_ptr<long>(),
            global_expert_count.data_ptr<long>(),
            d_model, d_hidden, num_expert, n_workers, has_bias,
            smgr);
    }));
    return {output_buf, global_input_buf, global_middle_buf, global_output_buf};
}

#endif

